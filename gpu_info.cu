
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("Device %d: %s\n", dev, deviceProp.name);
        printf("GPU Warp Size: %d\n", deviceProp.warpSize);
        printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("Maximum GPU Memory Size: %lu bytes\n", deviceProp.totalGlobalMem);
        printf("Maximum Block Dimensions: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("Maximum Grid Dimensions: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    }
    return 0;
}
